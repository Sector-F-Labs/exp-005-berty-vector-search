
#include <hip/hip_runtime.h>
extern "C" __global__ void cosine_similarity(
	const float* a,
	const float* b,
	float* dot_product,
	float* magnitude_a,
	float* magnitude_b,
	const int size
) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x; //global index
	if (index < size) {
		atomicAdd(dot_product, a[index] * b[index]);
		atomicAdd(magnitude_a, a[index] * a[index]);
		atomicAdd(magnitude_b, b[index] * b[index]);
	}
}
